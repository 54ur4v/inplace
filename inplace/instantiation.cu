#include "hip/hip_runtime.h"
#include "c2r.h"
#include "register_ops.h"
#include "sm.h"

#ifndef INSTANTIATED_TYPE
#define INSTANTIATED_TYPE double
#endif

#ifndef WPT
#define WPT 5
#endif

#ifndef SM
#define SM sm_20
#endif

namespace inplace {
namespace detail {

template __global__ void register_row_shuffle<SM, INSTANTIATED_TYPE, WPT>(int, int, INSTANTIATED_TYPE*, shuffle);

}
}
