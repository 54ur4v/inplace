#include "hip/hip_runtime.h"
#include "equations.h"
#include "register_ops.h"
#include "sm.h"

#ifndef INSTANTIATED_TYPE
#define INSTANTIATED_TYPE double
#endif

#ifndef WPT
#define WPT 5
#endif

#ifndef SM
#define SM sm_20
#endif

namespace inplace {
namespace detail {

template __global__ void register_row_shuffle<SM, INSTANTIATED_TYPE, c2r::shuffle, WPT>(int, int, INSTANTIATED_TYPE*, c2r::shuffle);

template __global__ void register_row_shuffle<SM, INSTANTIATED_TYPE, r2c::shuffle, WPT>(int, int, INSTANTIATED_TYPE*, r2c::shuffle);

}
}
