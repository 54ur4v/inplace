#include "hip/hip_runtime.h"
#include "permute.h"
#include <cstdio>
namespace inplace {
namespace detail {

struct scatter_permutes {
    typedef int result_type;
    int m; int n; int c; int a; int b; int q;
    __host__
    scatter_permutes(int _m, int _n, int _c) : m(_m), n(_n), c(_c) {
        int d;
        extended_gcd(n/c, m/c, d, q);
        a = m / c;
        b = n / c;
    }
    __host__ __device__
    int operator()(int i) const {
        int k = ((c - 1) * i) % c;
        int l = ((c - 1 + i) / c);
        int r = k * a + ((l * q) % a);
        return r;
    }
    __host__ __device__
    int len() const {
        return m;
    }
};
    
template<typename Fn>
void scatter_cycles(Fn f, std::vector<int>& heads, std::vector<int>& lens) {
    int len = f.len();
    thrust::counting_iterator<int> i(0);
    std::set<int> unvisited(i, i+len);
    while(!unvisited.empty()) {
        int idx = *unvisited.begin();
        unvisited.erase(unvisited.begin());
        int dest = f(idx);
        if (idx != dest) {
            heads.push_back(idx);
            int start = idx;
            int len = 1;
            //std::cout << "Cycle: " << start << " " << dest << " ";
            while(dest != start) {
                idx = dest;
                unvisited.erase(idx);
                dest = f(idx);
                len++;
                //std::cout << dest << " ";
            }
            //std::cout << std::endl;
            lens.push_back(len);
        }
    }
}


template<typename T, int U>
__device__ __forceinline__ void unroll_cycle_row_permute(
    scatter_permutes f, row_major_index rm, T* data, int i, int j, int l) {
    
    T src = data[rm(i, j)];
    T loaded[U+1];
    loaded[0] = src;
    for(int k = 0; k < l / U; k++) {
        int rows[U];
#pragma unroll
        for(int x = 0; x < U; x++) {
            i = f(i);
            rows[x] = i;
        }
#pragma unroll
        for(int x = 0; x < U; x++) {
            loaded[x+1] = data[rm(rows[x], j)];
        }
#pragma unroll
        for(int x = 0; x < U; x++) {
            data[rm(rows[x], j)] = loaded[x];
        }
        loaded[0] = loaded[U];
    }
    T tmp = loaded[0];
    // if (threadIdx.x == 0) {
    //     printf("Block: (%d, %d), Thread: (%d, %d), len: %d, U: %d\n",
    //            blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y,
    //            l, U);
    // }
    for(int k = 0; k < l % U; k++) {
        i = f(i);
        T new_tmp = data[rm(i, j)];
        data[rm(i, j)] = tmp;
        tmp = new_tmp;
    }

    
    
    // i = f(start);
    // T* src_ptr = data + rm(start, j);
    // T* dest_ptr = data + rm(i, j);
    // T src = *src_ptr;
    // do {
    //     T dest = *dest_ptr;
    //     *dest_ptr = src;
    //     src = dest;
    //     i = f(i);
    //     dest_ptr = data + rm(i, j);
    // } while(i != start);
    // *dest_ptr = src;
    
    // T src[U];
    // int inc = gridDim.x * blockDim.x;
    // int index = start;
    // T* src_ptr = data + rm(index, j);
    // #pragma unroll
    // for(int i = 0; i < U; i++) {
    //     src[i] = *src_ptr;
    //     src_ptr += inc;
    // }
    // do {
    //     index = f(index);
    //     src_ptr = data + rm(index, j);
    //     T dest[U];
    //     T* load_ptr = src_ptr;
    //     #pragma unroll
    //     for(int i = 0; i < U; i++) {
    //         dest[i] = *load_ptr;
    //         load_ptr += inc;
    //     }
    //     #pragma unroll
    //     for(int i = 0; i < U; i++) {
    //         *src_ptr = src[i];
    //         src[i] = dest[i];
    //         src_ptr += inc;
    //     }
    // } while (index != start);
}

template<typename T, int U>
__global__ void cycle_row_permute(scatter_permutes f, T* data, int* heads,
                                  int* lens, int n_heads) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int h = blockIdx.y * blockDim.y + threadIdx.y;
    int n = f.n;
    row_major_index rm(f.m, f.n);


    if ((j < n) && (h < n_heads)) {
        int i = heads[h];
        int l = lens[h];
        unroll_cycle_row_permute<T, U>(f, rm, data, i, j, l);
    }
}


template<typename T>
void postpermute(int m, int n, int c, T* data, int* tmp) {
    scatter_permutes f(m, n, c);
    std::vector<int> heads;
    std::vector<int> lens;
    scatter_cycles(f, heads, lens);
    int* d_heads = tmp;
    int* d_lens = tmp + m / 2;
    hipMemcpy(d_heads, heads.data(), sizeof(int)*heads.size(),
               hipMemcpyHostToDevice);
    hipMemcpy(d_lens, lens.data(), sizeof(int)*lens.size(),
               hipMemcpyHostToDevice);
    // std::ostream_iterator<int> os(std::cout, " ");
    // std::cout << "Heads: ";
    // std::copy(heads.begin(), heads.end(), os); std::cout << std::endl;
    // std::cout << "Lens: ";
    // std::copy(lens.begin(), lens.end(), os); std::cout << std::endl;

    int n_threads_x = 256;
    int n_threads_y = 1024/n_threads_x;
    
    int n_blocks_x = div_up(n, n_threads_x);
    int n_blocks_y = div_up(heads.size(), n_threads_y);
    cycle_row_permute<T, 4><<<dim3(n_blocks_x, n_blocks_y),
        dim3(n_threads_x, n_threads_y)>>>
        (f, data, d_heads, d_lens, heads.size());
    
}


template void postpermute<float>(int, int, int, float*, int*);
template void postpermute<double>(int, int, int, double*, int*);
template void postpermute<int>(int, int, int, int*, int*);


}
}
