#include "hip/hip_runtime.h"
#include "schedule.h"
#include "gcd.h"
#include "temporary.h"
#include "introspect.h"
#include "sm.h"
#include "rotate.h"
#include "permute.h"
#include "equations.h"
#include <algorithm>
#include <typeinfo>
#include <iostream>


namespace inplace {
namespace detail {


template<typename T, typename F>
__global__ void smem_row_shuffle(int m, int n, T* d, F s);

template<typename SM, typename T, typename F, int WPT>
__global__ void register_row_shuffle(int m, int n, T* d, F s);

template<typename T, typename F>
__global__ void memory_row_shuffle(int m, int n, T* d, T* tmp, F s);



template<typename T, typename Schedule, typename SM>
struct shuffle_enactor {};

template<typename T, typename SM, int blks>
struct shuffle_enactor<T, smem<T, SM, blks>, SM> {
    bool enabled;
    static const int blk = smem<T, SM, blks>::blk;
    static const int lim = smem<T, SM, blks>::lim;
    shuffle_enactor(int n) {
        enabled = (n <= lim);
    }
    template<typename F>
    void operator()(T* data, F s, temporary_storage<T> temp) {
        int smem_bytes = sizeof(T) * s.n;
        smem_row_shuffle<<<s.m, blk, smem_bytes>>>(s.m, s.n, data, s);
    }
};

template<typename T, typename SM, int w, int b>
struct shuffle_enactor<T, reg<w, b>, SM> {
    bool enabled;
    static const int wpt = reg<w, b>::wpt;
    static const int blk = reg<w, b>::blk;
    shuffle_enactor(int n) {
        enabled = (n <= reg<w, b>::lim);
    }
    template<typename F>
    void operator()(T* data, F s, temporary_storage<T> temp) {
        register_row_shuffle<SM, T, F, wpt>
            <<<s.m, blk>>>(s.m, s.n, data, s);
    }
};


template<typename T, typename SM>
struct shuffle_enactor<T, memory, SM> {
    bool enabled;
    shuffle_enactor(int n) {
        enabled = true;
    }
    template<typename F>
    void operator()(T* data, F s, temporary_storage<T> temp) {
        memory_row_shuffle
            <<<n_ctas(), n_threads()>>>(s.m, s.n, data, static_cast<T*>(temp), s);
    }
};

template<typename SM, typename T, typename F, typename Schedule, template<class, class, class> class Enactor>
struct enact_schedule {
    static void impl(T* data, F s, temporary_storage<T> temp) {
        Enactor<T, typename Schedule::head, SM>
            enactor(s.n);
        if (enactor.enabled) {
            enactor(data, s, temp);
        } else {
            enact_schedule<SM, T, F, typename Schedule::tail, Enactor>
                ::impl(data, s, temp);
        }
    }
};

template<typename SM, typename T, typename F, template<class, class, class> class Enactor>
struct enact_schedule<SM, T, F, memory, Enactor> {
    static void impl(T* data, F s, temporary_storage<T> temp) {
        Enactor<T, memory, SM> enactor(s.n);
        enactor(data, s, temp);
    }
};


template<typename T, typename F>
void shuffle_fn(T* data, F s, temporary_storage<T> temp) {
    int arch = current_sm();
    if (arch >= 305) {
        enact_schedule<sm_35, T, F, typename schedule<T, sm_35>::type, shuffle_enactor>
            ::impl(data, s, temp);
    } else if (arch >= 200) {
        enact_schedule<sm_20, T, F, typename schedule<T, sm_20>::type, shuffle_enactor>
            ::impl(data, s, temp);
    }
}

}

template<typename T>
void transpose_fn(bool row_major, T* data, int m, int n, T* tmp) {
    if (!row_major) {
        std::swap(m, n);
    }
    std::cout << "Doing transpose of " << m << ", " << n << std::endl;
    temporary_storage<T> temp_storage(m, n, tmp);

    int c, t, k;
    extended_gcd(m, n, c, t);
    if (c > 1) {
        extended_gcd(m/c, n/c, t, k);
    } else {
        k = t;
    }
    if (c > 1) {
        detail::prerotate(c, m, n, data);
    }
    detail::shuffle_fn(data, detail::c2r::shuffle(m, n, c, k), temp_storage);
    detail::postrotate(m, n, data);
    int* temp_int = (int*)(static_cast<T*>(temp_storage));
    detail::postpermute(m, n, c, data, temp_int);
}


void transpose(bool row_major, float* data, int m, int n, float* tmp) {
    transpose_fn(row_major, data, m, n, tmp);
}
void transpose(bool row_major, double* data, int m, int n, double* tmp) {
    transpose_fn(row_major, data, m, n, tmp);
}

}
