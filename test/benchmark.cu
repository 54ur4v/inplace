#include "hip/hip_runtime.h"
#include <iostream>
#include "transpose.h"

#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/functional.h>
#include <cstdlib>
#include "util.h"

using namespace inplace;


void visual_test(int m, int n) {
    thrust::device_vector<float> x(m*n);
    thrust::counting_iterator<int> c(0);
    thrust::transform(c, c+(m*n), x.begin(), thrust::identity<int>());
    print_array(x, row_major_index(m, n));
    c2r::transpose(true, thrust::raw_pointer_cast(x.data()), m, n);
    std::cout << std::endl;
    //print_array(x, row_major_index(m, n));
    print_array(x, row_major_index(n, m));
}


template<typename T>
void time_test(int m, int n) {
    bool row_major = true;//rand() & 2;

    std::cout << "Checking results for transpose of a " << m << " x " <<
        n << " matrix, in ";
    if (row_major) {
        std::cout << "row major order." << std::endl;
    } else {
        std::cout << "column major order." << std::endl;
    }
    
    thrust::device_vector<T> x(m*n);
    thrust::counting_iterator<int> c(0);
    thrust::transform(c, c+(m*n), x.begin(), thrust::identity<T>());
    //Preallocate temporary storage.
    thrust::device_vector<T> t(max(m,n)*n_ctas());
    hipEvent_t start,stop;
    float time=0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    
    inplace::r2c::transpose(row_major,
                            thrust::raw_pointer_cast(x.data()),
                            m, n);


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    std::cout << "  Time: " << time << " ms" << std::endl;
    float gbs = (float)(2 * m * n * sizeof(T)) / (time * 1000000);
    std::cout << "  Throughput: " << gbs << " GB/s" << std::endl;

    
    bool correct;
    if (row_major) {
        correct = is_tx_row_major(x, m, n);
    } else {
        correct = is_tx_col_major(x, m, n);
    }
    if (correct) {
        std::cout << "PASSES" << std::endl << std::endl;
    } else {
        std::cout << "FAILS" << std::endl << std::endl;
        exit(2);
    }
}

void generate_random_size(int& m, int &n) {
    size_t memory_size = gpu_memory_size();
    size_t ints_size = memory_size / sizeof(int);
    size_t e = 29440;//(size_t)sqrt(double(ints_size));
    while(true) {
        long long lm = 32 + rand() % e;
        long long ln = 32 + rand() % e;
        size_t extra = n_ctas() * max(lm, ln);
        if ((lm * ln > 0) && ((lm * (ln + extra)) < ints_size)) {
            m = (int)lm;
            n = (int)ln;
            return;
        }
    }
}

int main() {
    // for(int m = 32; m < 1000; m++) {
    //     for(int n = 1; n < 1000; n++) {
    //         time_test<double>(m, n);
    //     }
    // }
    //visual_test(32, 6);
    // time_test<double>(32, 6);
    // time_test<double>(13985, 512);
    // for(int i = 0; i < 1000; i++) {
    //    int m, n;
    //    generate_random_size(m, n);
    //    time_test<double>(m, n);
    // }
    time_test<double>(13986, 512);
    // int n_pts = 1000;
    // int l_bound = 1000;
    // int u_bound = 20000;
    // int delta = (u_bound - l_bound) / n_pts;
    // for(int m = l_bound; m < u_bound; m += delta) {
    //     for(int n = l_bound; n < u_bound; n += delta) {
    //         time_test<double>(m, n);
    //     }
    // }
        
}
